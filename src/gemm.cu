#include <stdlib.h>
#include <iostream>
#include <stdio.h>
#include <hipblas.h>
#include <cstdio>
#include <cstdlib>
#include "trans-conv.h"

void blas_gemm(const float *A, const float *B, float *C, const int m, const int k, const int n) {
    // define hipblasSgemm parameters
    int lda=m,ldb=k,ldc=m;
    const float alf = 1;
    const float bet = 0;
    const float *alpha = &alf;
    const float *beta = &bet;

    // create handle
    hipblasHandle_t handle;
    hipblasCreate(&handle); 

    // call gemm
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);

    // destory handle
    hipblasDestroy(handle);
}

bool gemm(float *opera, float *operb, float *res, int H, int W, int K){
    // define input and output dimensions
    int i, rows_A, cols_A, rows_B, cols_B, rows_C, cols_C;
    rows_A = rows_C = H;
    cols_A = rows_B = K;
    cols_B = cols_C = W;

    // allocate device memories
    float *device_A, *device_B, *device_C;
    hipMalloc(&device_A, rows_A * cols_A * sizeof(float));
    hipMalloc(&device_B, rows_B * cols_B * sizeof(float));
    hipMalloc(&device_C, rows_C * cols_C * sizeof(float));

    // set the values of device matrices
    hipblasStatus_t status;
    status = hipblasSetMatrix(rows_A, cols_A, sizeof(float), opera, rows_A, device_A, rows_A);
    if (status != HIPBLAS_STATUS_SUCCESS) 
    {
        throw EXIT_FAILURE;
    }

    status = hipblasSetMatrix(rows_B, cols_B, sizeof(float), operb, rows_B, device_B, rows_B);
    if (status != HIPBLAS_STATUS_SUCCESS) 
    {
        throw EXIT_FAILURE;
    }

    // Multiply A and B on GPU
    blas_gemm(device_A, device_B, device_C, rows_A, cols_A, cols_B);

    // Copy (and print) the result on host memory
    hipMemcpy(res, device_C, rows_C * cols_C * sizeof(float), hipMemcpyDeviceToHost);

    //Free GPU memory
    hipFree(device_A);
    hipFree(device_B);
    hipFree(device_C);

    return 0;
}